#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <opencv2/opencv.hpp>
#include <cfloat>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/border_interpolate.hpp>
#include <opencv2/core/cuda/vec_traits.hpp>
#include <opencv2/core/cuda/vec_math.hpp>

#define TILE_SIZE 15

/**
 * Clamps the value val in the interval [lo, high].
 * Equivalent to max(lo, min(val, high)).
 *
 * @param val: value to clamp.
 * @param lo: lower bound for the clamping.
 * @param high: higher bound for the clamping.
 * @return val clamped between lo and high.
 */
template< typename T > __device__ T clamp(T val, T lo, T high) {
  return max(lo, min(val, high));
}

/**
 * Returns the value of a Gaussian function with an standard
 * deviation of sigma and an input value of x.
 *
 * @param x: input value of the Gaussian function.
 * @param sigma: standard deviation of Gaussian function.
 * @return result of the Gaussian function.
 */
__device__ float gauss_func(int x, float sigma) {
  return exp(-(pow(x, 2.0) / (2.0 * pow(sigma, 2.0))));
}

__global__ void process(const cv::cuda::PtrStep<uchar3> src,
                        cv::cuda::PtrStep<uchar3> dst, int rows, int cols,
                        int kernel_size, int sigma, bool first_pass) {

    const int dst_x = TILE_SIZE * blockIdx.x + threadIdx.x-kernel_size;
    const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

    // Filter radius
    const int kernel_div2 = kernel_size / 2;

    // Create shared memory using externally passed size
    extern __shared__ uchar3 tile[];

    int px = clamp<float>(dst_x, 0, cols-1);
    int py = clamp<float>(dst_y, 0, rows-1);

    // Cache pixels in shared memory
    tile[threadIdx.x] = src(py, px);

    // Wait until all thread cache their pixes values
    __syncthreads();  

    bool is_inside_tile =
        kernel_div2 <= threadIdx.x && threadIdx.x < TILE_SIZE + kernel_div2;
    if (dst_x < cols && dst_y < rows && is_inside_tile) {
	float3 val = make_float3(0, 0, 0);
	float gauss_sum = 0;
	for (int m = -kernel_div2; m <= kernel_div2; m++) {
	    float gauss_val = gauss_func(m, sigma);
	    gauss_sum += gauss_val;

	    int tx = threadIdx.x+m;
	    uchar3 pix = tile[tx];
	    val.x += (float)pix.x*gauss_val;
	    val.y += (float)pix.y*gauss_val;
	    val.z += (float)pix.z*gauss_val;
	}
	val.x = val.x/gauss_sum;
	val.y = val.y/gauss_sum;
	val.z = val.z/gauss_sum;

        dst(dst_y, dst_x).x = val.x;
        dst(dst_y, dst_x).y = val.y;
        dst(dst_y, dst_x).z = val.z;
    }
}

int divUp(int a, int b) {
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void startCUDA (cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, int KERNEL_SIZE,
                float SIGMA, bool first_pass) {
  const dim3 block(TILE_SIZE+KERNEL_SIZE);
  const dim3 grid(divUp(dst.cols, TILE_SIZE)+1, divUp(dst.rows, block.y));

  
  // Create a tile to process pixels within a block's shared memory
  int shmem_size = sizeof(uchar3)*(TILE_SIZE+KERNEL_SIZE);
  
  process<<<grid, block, shmem_size>>>(src, dst, dst.rows, dst.cols,
				       KERNEL_SIZE, SIGMA, first_pass);

}

